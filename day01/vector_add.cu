#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

// Error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

int main(void) {
    // Vector size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, size));

    // Copy input vectors from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Check for kernel launch errors
    CHECK_CUDA_ERROR(hipGetLastError());

    // Wait for kernel to finish
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Now safe to copy results back to host
    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Verify the result
    float maxError = 0.0f;
    for (int i = 0; i < numElements; i++) {
        maxError = fmax(maxError, fabs(h_C[i] - (h_A[i] + h_B[i])));
    }
    printf("Max error: %f\n", maxError);

    // Free device memory
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Vector addition completed successfully!\n");
    return 0;
}
